#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <cfloat>
#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <sstream>
#include <cmath>
#include <algorithm>
#include <cfloat>  // For DBL_MAX
#include <iomanip> // For std::setprecision
#include <chrono>  // For timing
#include <omp.h>
// For DBL_MAX

// CUDA specific headers
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h> // Required for grid.sync()

namespace cg = cooperative_groups;

// ===================================================================================
// KERNEL-RELATED DATA STRUCTURES & DEFINITIONS
// ===================================================================================

// Use standard integer and double types for clarity
using namespace std;

//~ Define types
using point_t = double;
using weight_t = double;
using demand_t = double;
using node_t = int; // let's keep as int than unsigned. -1 is init. nodes ids 0 to n-1

const node_t DEPOT = 0;
// A simple struct for 3D points/nodes.
// This must be a POD (Plain Old Data) type for easy GPU memory copies.
struct Point
{
    double x, y, demand;
};

// The output struct that the kernel will populate.
// 'volatile' is used to advise the compiler against certain optimizations
// and ensure reads/writes happen as expected in a parallel context.
struct Saving
{
    volatile int i, j;
    volatile long long value;
};

class VRP
{
public:
    size_t size;
    demand_t capacity;
    std::vector<Point> node;
    std::vector<weight_t> dist_to_depot;

    VRP() : size(0), capacity(0) {}

    void read(const std::string &filename);
    weight_t get_dist(node_t i, node_t j) const;

    size_t getSize() const
    {
        return size;
    }
    demand_t getCapacity() const
    {
        return capacity;
    }
};

void VRP::read(const std::string &filename)
{
    std::ifstream in(filename);
    if (!in.is_open())
    {
        std::cerr << "Error: Could not open file " << filename << std::endl;
        exit(1);
    }
    std::string line;
    while (getline(in, line) && line.find("DIMENSION") == std::string::npos)
        ;
    if (line.find(":") != std::string::npos)
        size = stoul(line.substr(line.find(":") + 1));
    while (getline(in, line) && line.find("CAPACITY") == std::string::npos)
        ;
    if (line.find(":") != std::string::npos)
        capacity = stoul(line.substr(line.find(":") + 1));
    while (getline(in, line) && line.find("NODE_COORD_SECTION") == std::string::npos)
        ;
    node.resize(size);
    for (size_t i = 0; i < size; ++i)
    {
        int id;
        in >> id >> node[i].x >> node[i].y;
    }
    while (getline(in, line) && line.find("DEMAND_SECTION") == std::string::npos)
        ;
    for (size_t i = 0; i < size; ++i)
    {
        int id;
        in >> id >> node[i].demand;
    }
    in.close();
    dist_to_depot.resize(size);
    for (size_t i = 0; i < size; ++i)
    {
        dist_to_depot[i] = get_dist(DEPOT, i);
    }
}

weight_t VRP::get_dist(node_t i, node_t j) const
{
    double dx = node[i].x - node[j].x;
    double dy = node[i].y - node[j].y;
    return sqrt(dx * dx + dy * dy);
}

/**
 * @brief Calculates the total travel cost for a set of routes.
 * @param vrp The VRP instance.
 * @param routes A vector of routes, where each route is a vector of customer nodes.
 * @return The total Euclidean distance for all routes, including travel from and to the depot.
 */
weight_t calCost(const VRP &vrp, const std::vector<std::vector<node_t>> &routes)
{
    weight_t total_cost = 0.0;
    for (const auto &route : routes)
    {
        if (route.empty())
            continue;

        node_t last_node = DEPOT;
        for (node_t current_node : route)
        {
            total_cost += vrp.get_dist(last_node, current_node);
            last_node = current_node;
        }
        // Add cost to return to the depot
        total_cost += vrp.get_dist(last_node, DEPOT);
    }
    return total_cost;
}

/**
 * @brief Verifies that the solution is valid by checking capacity constraints.
 * @param vrp The VRP instance.
 * @param routes The solution routes to verify.
 * @return True if all routes respect the vehicle capacity, false otherwise.
 */
bool verify_sol(const VRP &vrp, const std::vector<std::vector<node_t>> &routes)
{
    for (const auto &route : routes)
    {
        demand_t route_demand = 0;
        for (node_t customer : route)
        {
            if (customer < 0 || customer >= vrp.size)
                return false; // Invalid node
            route_demand += vrp.node[customer].demand;
        }
        if (route_demand > vrp.getCapacity())
        {
            return false; // Capacity violated
        }
    }
    return true;
}

// ===================================================================================
// CUDA ERROR CHECKING UTILITY (Essential for debugging)
// ===================================================================================
#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA Error at " << file << ":" << line << " code=" << static_cast<unsigned int>(result)
                  << " \"" << hipGetErrorString(result) << "\" for " << func << std::endl;
        hipDeviceReset();
        exit(99);
    }
}

// ===================================================================================
// DEVICE (GPU) UTILITY FUNCTION
// ===================================================================================

// __device__ function can only be called from other GPU functions (e.g., the kernel)
__device__ double device_euclidean_dist(const Point &a, const Point &b)
{
    return sqrt((a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y));
}

// ===================================================================================
// THE COMPLETE CUDA KERNEL
// ===================================================================================

/**
 * @brief CUDA kernel to find the single best saving among all valid customer pairs.
 *
 * This kernel launches a 2D grid where each thread handles one pair (i, j).
 * It performs validity checks (route, capacity, endpoints) and calculates the saving.
 * A robust atomic reduction using a compare-and-swap loop finds the global maximum
 * saving across all threads and updates the single output struct.
 *
 * @param nodes              Device pointer to array of all nodes (depot at index 0).
 * @param customer_route_map Device pointer mapping customer ID to its route ID.
 * @param route_demands      Device pointer to an array of current demands for each route.
 * @param route_endpoints    Device pointer storing the front and back customer for each route.
 * @param best_saving_out    Device pointer to a single Saving struct to store the result.
 * @param num_customers      The total number of customers (problem size 'n').
 * @param capacity           The vehicle capacity.
 */
__global__ void find_best_saving_kernel(
    const Point *nodes,
    const node_t *customer_route_map,
    const demand_t *route_demands,
    const node_t *route_endpoints,
    Saving *best_saving_out,
    int num_customers,
    demand_t capacity)
{
    // --- 1. Map thread to a unique customer pair (i, j) ---
    // Customer IDs are 1-based, so we add 1 to the thread's calculated index.
    cg::grid_group grid = cg::this_grid();
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    long long saving_value = 0;

    // Exit if thread is outside the problem space or in the lower triangle of the pair matrix.
    if (i >= (num_customers + 1) || j >= (num_customers + 1) || i >= j)
    {
    }
    else
    {
        node_t route_id_i = customer_route_map[i];
        node_t route_id_j = customer_route_map[j];
        if ((route_id_i == route_id_j || route_id_i < 0 || route_id_j < 0) || (route_demands[route_id_i] + route_demands[route_id_j] > capacity))
        {
        }
        else
        {
            node_t front_i = route_endpoints[route_id_i * 2];
            node_t back_i = route_endpoints[route_id_i * 2 + 1];
            node_t front_j = route_endpoints[route_id_j * 2];
            node_t back_j = route_endpoints[route_id_j * 2 + 1];
            if (!((i == front_i || i == back_i) && (j == front_j || j == back_j)))
            {
            }
            else
            {
                saving_value = round(device_euclidean_dist(nodes[i], nodes[0])     // dist(i, depot)
                                     + device_euclidean_dist(nodes[j], nodes[0])   // dist(j, depot)
                                     - device_euclidean_dist(nodes[i], nodes[j])); // dist(i, j)
                unsigned long long int *global_max_addr_ull = (unsigned long long int *)&(best_saving_out->value);

                atomicMax(global_max_addr_ull, saving_value);
            }
        }
    }

    grid.sync();
    if (saving_value == best_saving_out->value)
    {

        atomicMin((unsigned int *)&(best_saving_out->i), (unsigned int)i);
    }
    grid.sync();
    if (best_saving_out->i == (unsigned int)i && saving_value == best_saving_out->value)
    {
        best_saving_out->j = (unsigned int)j;
    }
}

std::vector<std::vector<node_t>> parallel_savings_algorithm(const VRP &vrp)
{
    int NUM_CUSTOMERS = vrp.getSize() - 1; // Exclude depot
    demand_t CAPACITY = vrp.getCapacity();

    std::vector<node_t> h_customer_route_map(NUM_CUSTOMERS + 1);
    std::vector<demand_t> h_route_demands(NUM_CUSTOMERS + 1);
    std::vector<node_t> h_route_endpoints((NUM_CUSTOMERS + 1) * 2);

    for (int i = 1; i <= NUM_CUSTOMERS; ++i)
    {
        // Initially, each customer is in their own route
        h_customer_route_map[i] = i;
        h_route_demands[i] = vrp.node[i].demand;
        // The start and end of the route is just the customer itself
        h_route_endpoints[i * 2] = i;     // Front
        h_route_endpoints[i * 2 + 1] = i; // Back
    }

    // --- 2. DEVICE: Allocate GPU memory ---
    Point *d_nodes;
    node_t *d_customer_route_map;
    demand_t *d_route_demands;
    node_t *d_route_endpoints;
    Saving *d_best_saving_out;

    checkCudaErrors(hipMalloc(&d_nodes, (NUM_CUSTOMERS + 1) * sizeof(Point)));
    checkCudaErrors(hipMalloc(&d_customer_route_map, (NUM_CUSTOMERS + 1) * sizeof(node_t)));
    checkCudaErrors(hipMalloc(&d_route_demands, (NUM_CUSTOMERS + 1) * sizeof(demand_t)));
    checkCudaErrors(hipMalloc(&d_route_endpoints, (NUM_CUSTOMERS + 1) * 2 * sizeof(node_t)));
    checkCudaErrors(hipMalloc(&d_best_saving_out, sizeof(Saving)));

    // --- 3. HOST -> DEVICE: Copy data to GPU ---
    checkCudaErrors(hipMemcpy(d_nodes, vrp.node.data(), (NUM_CUSTOMERS + 1) * sizeof(Point), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_customer_route_map, h_customer_route_map.data(), (NUM_CUSTOMERS + 1) * sizeof(node_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_route_demands, h_route_demands.data(), (NUM_CUSTOMERS + 1) * sizeof(demand_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_route_endpoints, h_route_endpoints.data(), (NUM_CUSTOMERS + 1) * 2 * sizeof(node_t), hipMemcpyHostToDevice));

    // Initialize the output struct on the GPU to a known "worst" state
    Saving h_best_saving_init = {-1, -1, 0};
    checkCudaErrors(hipMemcpy(d_best_saving_out, &h_best_saving_init, sizeof(Saving), hipMemcpyHostToDevice));

    // --- 4. KERNEL LAUNCH ---
    dim3 threadsPerBlock(128, 128);
    dim3 numBlocks(
        (NUM_CUSTOMERS + 1 + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (NUM_CUSTOMERS + 1 + threadsPerBlock.y - 1) / threadsPerBlock.y);

    void *args[] = {
        &d_nodes,
        &d_customer_route_map,
        &d_route_demands,
        &d_route_endpoints,
        &d_best_saving_out,
        (void *)&NUM_CUSTOMERS, // Must cast pointers to void*
        &CAPACITY};

    // --- 3. Initialize Routes (Sequential) ---
    std::vector<std::vector<node_t>> routes(vrp.size);
    for (node_t i = 1; i < vrp.size; ++i)
    {
        routes[i] = {i};
    }
    Saving h_result;

    // --- 4. Merge Routes Greedily (Sequential) ---
    int id = 0;
    while (true)
    {
        // std::cout << id++ << "\n";
        if (id > NUM_CUSTOMERS)
            break;
        id++;
        // std::cout << id << "\n";
        checkCudaErrors(hipMemcpy(d_customer_route_map, h_customer_route_map.data(), (NUM_CUSTOMERS + 1) * sizeof(node_t), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_route_demands, h_route_demands.data(), (NUM_CUSTOMERS + 1) * sizeof(demand_t), hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(d_route_endpoints, h_route_endpoints.data(), (NUM_CUSTOMERS + 1) * 2 * sizeof(node_t), hipMemcpyHostToDevice));

        checkCudaErrors(hipMemcpy(d_best_saving_out, &h_best_saving_init, sizeof(Saving), hipMemcpyHostToDevice));
        checkCudaErrors(hipLaunchCooperativeKernel(
            (void *)find_best_saving_kernel,
            numBlocks,
            threadsPerBlock,
            args));
        checkCudaErrors(hipDeviceSynchronize());

        checkCudaErrors(hipMemcpy(&h_result, d_best_saving_out, sizeof(Saving), hipMemcpyDeviceToHost));
        if (h_result.value <= 1e-6)
        {
            std::cout << "No more positive savings found. Halting." << std::endl;
            std::cout << id << "\n";
            break; // Exit the while loop
        }

        node_t i = h_result.i;
        node_t j = h_result.j;
        // std::cout << i << " " << j << "\n"

        node_t route_id_i = h_customer_route_map[i];
        node_t route_id_j = h_customer_route_map[j];

        // Check if the merge is valid (different routes and combined demand is within capacity)
        if (route_id_i != route_id_j && h_route_demands[route_id_i] + h_route_demands[route_id_j] <= vrp.capacity)
        {
            // Determine the position of the merge nodes (i and j) in their respective routes
            bool i_is_front = (routes[route_id_i].front() == i);
            bool i_is_back = (routes[route_id_i].back() == i);
            bool j_is_front = (routes[route_id_j].front() == j);
            bool j_is_back = (routes[route_id_j].back() == j);

            node_t merged_into_route = -1;
            node_t consumed_route = -1;

            // Case 1: Merge End of route I -> Start of route J
            if (i_is_back && j_is_front)
            {
                routes[route_id_i].insert(routes[route_id_i].end(), routes[route_id_j].begin(), routes[route_id_j].end());
                merged_into_route = route_id_i;
                consumed_route = route_id_j;
            }
            // Case 2: Merge End of route J -> Start of route I
            else if (j_is_back && i_is_front)
            {
                routes[route_id_j].insert(routes[route_id_j].end(), routes[route_id_i].begin(), routes[route_id_i].end());
                merged_into_route = route_id_j;
                consumed_route = route_id_i;
            }
            // Case 3: Merge Start of route I -> Start of route J (requires reversing route I)
            else if (i_is_front && j_is_front)
            {
                std::reverse(routes[route_id_i].begin(), routes[route_id_i].end());
                routes[route_id_i].insert(routes[route_id_i].end(), routes[route_id_j].begin(), routes[route_id_j].end());
                merged_into_route = route_id_i;
                consumed_route = route_id_j;
            }
            // Case 4: Merge End of route I -> End of route J (requires reversing route J)
            else if (i_is_back && j_is_back)
            {
                std::reverse(routes[route_id_j].begin(), routes[route_id_j].end());
                routes[route_id_i].insert(routes[route_id_i].end(), routes[route_id_j].begin(), routes[route_id_j].end());
                merged_into_route = route_id_i;
                consumed_route = route_id_j;
            }

            // If a merge happened, update all related data structures
            if (merged_into_route != -1 && consumed_route != -1)
            {
                // 1. Update total demand of the merged route
                h_route_demands[merged_into_route] += h_route_demands[consumed_route];
                h_route_demands[consumed_route] = 0;

                // 2. Update the customer-to-route map for all nodes in the consumed route
                for (node_t customer : routes[consumed_route])
                {
                    h_customer_route_map[customer] = merged_into_route;
                }

                // 3. Update the endpoints of the newly formed merged route
                h_route_endpoints[merged_into_route * 2] = routes[merged_into_route].front();
                h_route_endpoints[merged_into_route * 2 + 1] = routes[merged_into_route].back();

                // 4. Invalidate the endpoints of the consumed route so the kernel ignores it
                h_route_endpoints[consumed_route * 2] = -1;
                h_route_endpoints[consumed_route * 2 + 1] = -1;

                // 5. Clear the vector of the consumed route
                routes[consumed_route].clear();
            }
        }
    }

    // --- 5. Finalize Routes ---
    std::vector<std::vector<node_t>> final_routes;
    for (const auto &route : routes)
    {
        if (!route.empty())
        {
            final_routes.push_back(route);
        }
    }

    checkCudaErrors(hipFree(d_nodes));
    checkCudaErrors(hipFree(d_customer_route_map));
    checkCudaErrors(hipFree(d_route_demands));
    checkCudaErrors(hipFree(d_route_endpoints));
    checkCudaErrors(hipFree(d_best_saving_out));
    checkCudaErrors(hipDeviceReset());

    return final_routes;
}

void tsp_approx(const VRP &vrp, std::vector<node_t> &cities, std::vector<node_t> &tour, node_t ncities)
{
    node_t i, j;
    node_t ClosePt = 0;
    weight_t CloseDist;

    for (i = 1; i < ncities; i++)
        tour[i] = cities[i - 1];

    tour[0] = cities[ncities - 1];

    for (i = 1; i < ncities; i++)
    {
        weight_t ThisX = vrp.node[tour[i - 1]].x;
        weight_t ThisY = vrp.node[tour[i - 1]].y;
        CloseDist = DBL_MAX;
        for (j = ncities - 1;; j--)
        {
            weight_t ThisDist = (vrp.node[tour[j]].x - ThisX) * (vrp.node[tour[j]].x - ThisX);
            if (ThisDist <= CloseDist)
            {
                ThisDist += (vrp.node[tour[j]].y - ThisY) * (vrp.node[tour[j]].y - ThisY);
                if (ThisDist <= CloseDist)
                {
                    if (j < i)
                        break;
                    CloseDist = ThisDist;
                    ClosePt = j;
                }
            }
        }
        unsigned temp = tour[i];
        tour[i] = tour[ClosePt];
        tour[ClosePt] = temp;
    }
}

std::vector<std::vector<node_t>>
postprocess_tsp_approx(const VRP &vrp, std::vector<std::vector<node_t>> &solRoutes)
{
    std::vector<std::vector<node_t>> modifiedRoutes;

    unsigned nroutes = solRoutes.size();
    for (unsigned i = 0; i < nroutes; ++i)
    {
        unsigned sz = solRoutes[i].size();
        std::vector<node_t> cities(sz + 1);
        std::vector<node_t> tour(sz + 1);

        for (unsigned j = 0; j < sz; ++j)
            cities[j] = solRoutes[i][j];

        cities[sz] = 0;

        tsp_approx(vrp, cities, tour, sz + 1);

        vector<node_t> curr_route;
        for (unsigned kk = 1; kk < sz + 1; ++kk)
        {
            curr_route.push_back(tour[kk]);
        }

        modifiedRoutes.push_back(curr_route);
    }
    return modifiedRoutes;
}

void tsp_2opt(const VRP &vrp, std::vector<node_t> &cities, std::vector<node_t> &tour, unsigned ncities)
{
    unsigned improve = 0;

    while (improve < 2)
    {
        double best_distance = 0.0;

        best_distance += vrp.get_dist(DEPOT, cities[0]);

        for (unsigned jj = 1; jj < ncities; ++jj)
        {
            best_distance += vrp.get_dist(cities[jj - 1], cities[jj]);
        }

        best_distance += vrp.get_dist(DEPOT, cities[ncities - 1]);

        for (unsigned i = 0; i < ncities - 1; i++)
        {
            for (unsigned k = i + 1; k < ncities; k++)
            {
                double new_distance = best_distance;
                if (i == 0)
                    new_distance -= vrp.get_dist(DEPOT, cities[i]);
                else
                    new_distance -= vrp.get_dist(cities[i - 1], cities[i]);

                if (k == ncities - 1)
                    new_distance -= vrp.get_dist(cities[k], DEPOT);
                else
                    new_distance -= vrp.get_dist(cities[k], cities[k + 1]);

                if (i == 0)
                    new_distance += vrp.get_dist(DEPOT, cities[k]);
                else
                    new_distance += vrp.get_dist(cities[i - 1], cities[k]);

                if (k == ncities - 1)
                    new_distance += vrp.get_dist(cities[i], DEPOT);
                else
                    new_distance += vrp.get_dist(cities[i], cities[k + 1]);

                if (new_distance < best_distance)
                {
                    improve = 0;
                    int left_ptr = i, right_ptr = k;
                    while (left_ptr <= right_ptr)
                    {
                        swap(cities[left_ptr++], cities[right_ptr--]);
                    }
                    best_distance = new_distance;
                }
            }
        }
        improve++;
    }
}
std::vector<std::vector<node_t>>
postprocess_2OPT(const VRP &vrp, std::vector<std::vector<node_t>> &final_routes)
{
    std::vector<std::vector<node_t>> postprocessed_final_routes;

    unsigned nroutes = final_routes.size();
    for (unsigned i = 0; i < nroutes; ++i)
    {
        unsigned sz = final_routes[i].size();
        std::vector<node_t> cities(sz);
        std::vector<node_t> tour(sz);

        for (unsigned j = 0; j < sz; ++j)
            cities[j] = final_routes[i][j];

        vector<node_t> curr_route;

        if (sz > 2)
            tsp_2opt(vrp, cities, tour, sz);

        for (unsigned kk = 0; kk < sz; ++kk)
        {
            curr_route.push_back(cities[kk]);
        }

        postprocessed_final_routes.push_back(curr_route);
    }
    return postprocessed_final_routes;
}

weight_t get_total_cost_of_routes(const VRP &vrp, vector<vector<node_t>> &final_routes)
{
    weight_t total_cost = 0.0;
    for (unsigned ii = 0; ii < final_routes.size(); ++ii)
    {
        weight_t curr_route_cost = 0;
        curr_route_cost += vrp.get_dist(DEPOT, final_routes[ii][0]);
        for (unsigned jj = 1; jj < final_routes[ii].size(); ++jj)
        {
            curr_route_cost += vrp.get_dist(final_routes[ii][jj - 1], final_routes[ii][jj]);
        }
        curr_route_cost += vrp.get_dist(DEPOT, final_routes[ii][final_routes[ii].size() - 1]);

        total_cost += curr_route_cost;
    }

    return total_cost;
}

std::vector<std::vector<node_t>>
postProcessIt(const VRP &vrp, std::vector<std::vector<node_t>> &final_routes, weight_t &minCost)
{
    std::vector<std::vector<node_t>> postprocessed_final_routes;

    auto postprocessed_final_routes1 = postprocess_tsp_approx(vrp, final_routes);
    auto postprocessed_final_routes2 = postprocess_2OPT(vrp, postprocessed_final_routes1);
    auto postprocessed_final_routes3 = postprocess_2OPT(vrp, final_routes);

#pragma omp parallel for
    for (unsigned zzz = 0; zzz < final_routes.size(); ++zzz)
    {
        vector<node_t> postprocessed_route2 = postprocessed_final_routes2[zzz];
        vector<node_t> postprocessed_route3 = postprocessed_final_routes3[zzz];

        unsigned sz2 = postprocessed_route2.size();
        unsigned sz3 = postprocessed_route3.size();

        weight_t postprocessed_route2_cost = 0.0;
        postprocessed_route2_cost += vrp.get_dist(DEPOT, postprocessed_route2[0]);
        for (unsigned jj = 1; jj < sz2; ++jj)
        {
            postprocessed_route2_cost += vrp.get_dist(postprocessed_route2[jj - 1], postprocessed_route2[jj]);
        }
        postprocessed_route2_cost += vrp.get_dist(DEPOT, postprocessed_route2[sz2 - 1]);

        weight_t postprocessed_route3_cost = 0.0;
        postprocessed_route3_cost += vrp.get_dist(DEPOT, postprocessed_route3[0]);
        for (unsigned jj = 1; jj < sz3; ++jj)
        {
            postprocessed_route3_cost += vrp.get_dist(postprocessed_route3[jj - 1], postprocessed_route3[jj]);
        }
        postprocessed_route3_cost += vrp.get_dist(DEPOT, postprocessed_route3[sz3 - 1]);

        if (postprocessed_route3_cost > postprocessed_route2_cost)
        {
            postprocessed_final_routes.push_back(postprocessed_route2);
        }
        else
        {
            postprocessed_final_routes.push_back(postprocessed_route3);
        }
    }

    auto postprocessed_final_routes_cost = get_total_cost_of_routes(vrp, postprocessed_final_routes);

    minCost = postprocessed_final_routes_cost;

    return postprocessed_final_routes;
}

int main(int argc, char *argv[])
{
    if (argc < 2)
    {
        std::cerr << "Usage: " << argv[0] << " <filename.vrp> [num_threads]" << std::endl;
        return 1;
    }

    VRP vrp;
    vrp.read(argv[1]);

    auto start_time = std::chrono::high_resolution_clock::now();
    std::vector<std::vector<node_t>> routes = parallel_savings_algorithm(vrp);
    auto end_time = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = end_time - start_time;
    weight_t total_cost = calCost(vrp, routes);
    auto postRoutes = postProcessIt(vrp, routes, total_cost);
    total_cost = calCost(vrp, postRoutes);
    bool is_valid = verify_sol(vrp, postRoutes);

    std::cout << "--- Parallel Clarke & Wright Savings Algorithm ---" << std::endl;
    std::cout << "Problem File: " << argv[1] << std::endl;
    // std::cout << "Threads Used: " << omp_get_max_threads() << std::endl;
    std::cout << "--------------------------------------------------" << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Total Solution Cost: " << total_cost << std::endl;
    std::cout << " Total Routes Used:  " << postRoutes.size() << std::endl;
    std::cout << "Total Time Taken:    " << elapsed.count() << " seconds" << std::endl;
    std::cout << "Solution Validity:   " << (is_valid ? "VALID" : "INVALID") << std::endl;
    std::cout << "--------------------------------------------------" << std::endl;

    return 0;
}
