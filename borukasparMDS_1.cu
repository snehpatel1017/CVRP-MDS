//~~~START:Thu, 16-Jun-2022, 12:43:32 IST
// For GECCO'23 Submission.
// MODIFIED FOR CUDA ACCELERATION
// COMPILE WITH THE FOLLOWING COMMAND:
// nvcc -O3 -std=c++14 -gencode arch=compute_75,code=sm_75 borukasparMDS.cu -o borukasparMDS.out
// (change compute_75 and sm_75 to your GPU's architecture if it's not a Tesla T4, e.g., sm_86 for Ampere)
/*
 * Original CPU Authors:
 * Rajesh Pandian M | https://mrprajesh.co.in
 * Somesh Singh     | https://ssomesh.github.io
 * Rupesh Nasre     | www.cse.iitm.ac.in/~rupesh
 * N.S.Narayanaswamy| www.cse.iitm.ac.in/~swamy
 * MIT LICENSE
 */

#include <iostream>
#include <vector>
#include <set>
#include <algorithm>
#include <cfloat>
#include <climits>
#include <cstring>
#include <iomanip>
#include <fstream>
#include <deque>
#include <sstream> //stringstream
#include <numeric> // For std::iota
#include <cmath>   // For sqrt

#include <random>
#include <chrono> //timing CPU

// CUDA Includes
#include <hip/hip_runtime.h>

#define DEBUGCODE 0
#define DEBUG if (DEBUGCODE)

// CUDA Error Checking Macro
#define gpuErrchk(ans)                        \
    {                                         \
        gpuAssert((ans), __FILE__, __LINE__); \
    }
inline void
gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort)
            exit(code);
    }
}

using namespace std;

//~ Define types
using point_t = double;
using weight_t = double;
using demand_t = double;
using node_t = int;
using ull = unsigned long long;

const node_t DEPOT = 0;
const ull ULL_INF = ~0ULL;

// To store all cmd line params in one struct
class Params
{
public:
    Params()
    {
        toRound = 1;
        nThreads = 20;
    }
    ~Params() {}
    bool toRound;
    short nThreads;
};

class Edge
{
public:
    node_t to;
    weight_t length;

    Edge() {}
    ~Edge() {}
    Edge(node_t t, weight_t l) : to(t), length(l) {}
    bool operator<(const Edge &e) { return length < e.length; }
};

class Point
{
public:
    point_t x;
    point_t y;
    demand_t demand;
};

// To Hold the contents input.vrp
class VRP
{
    demand_t capacity;
    string type;

public:
    size_t size;
    VRP() {}
    ~VRP() {}
    unsigned read(string filename);
    void print();
    weight_t get_dist(node_t i, node_t j, bool isRound = true) const
    {
        if (i == j)
            return 0.0;
        weight_t w = sqrt(pow(node[i].x - node[j].x, 2) + pow(node[i].y - node[j].y, 2));
        if (isRound == false)
            return w;
        return (params.toRound ? round(w) : w);
    }
    vector<Point> node;
    Params params;
    size_t getSize() const { return size; }
    demand_t getCapacity() const { return capacity; }
};

unsigned VRP::read(string filename)
{
    ifstream in(filename);
    if (!in.is_open())
    {
        cerr << "Could not open the file \"" << filename << "\"" << endl;
        exit(1);
    }
    string line;
    for (int i = 0; i < 3; ++i)
        getline(in, line);
    getline(in, line);
    size = stof(line.substr(line.find(":") + 2));
    getline(in, line);
    type = line.substr(line.find(":") + 2);
    getline(in, line);
    capacity = stof(line.substr(line.find(":") + 2));
    getline(in, line);
    node.resize(size);
    for (size_t i = 0; i < size; ++i)
    {
        getline(in, line);
        stringstream iss(line);
        size_t id;
        iss >> id >> node[i].x >> node[i].y;
    }
    getline(in, line);
    for (size_t i = 0; i < size; ++i)
    {
        getline(in, line);
        stringstream iss(line);
        size_t id;
        iss >> id >> node[i].demand;
    }
    in.close();
    return capacity;
}

void VRP::print()
{
    cout << "DIMENSION:" << size << '\n';
    cout << "CAPACITY:" << capacity << '\n';
    for (auto i = 0u; i < size; ++i)
    {
        cout << i << ':' << setw(6) << node[i].x << ' '
             << setw(6) << node[i].y << ' '
             << setw(6) << node[i].demand << endl;
    }
}

// START: CUDA BORUVKA'S ALGORITHM IMPLEMENTATION
// Simplified struct for MST edges on GPU, now stores indices instead of full objects
struct MSTEdge
{
    node_t u, v;
};

__device__ int find_repres(volatile int *comp, int v)
{
    int p = comp[v];
    if (v == p)
        return v;
    int gp = comp[p];
    while (p != gp)
    {
        atomicCAS((int *)&comp[v], p, gp);
        v = gp;
        p = comp[v];
        gp = comp[p];
    }
    return p;
}

__global__ void initialize_kernel(int V, ull *d_cheapest)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < V)
    {
        d_cheapest[tid] = ULL_INF;
    }
}

__global__ void find_cheapest_kernel(int V, const point_t *d_xs, const point_t *d_ys, volatile int *d_comp, ull *d_cheapest)
{
    int u = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = gridDim.x * blockDim.x;

    for (int i = u; i < V; i += gridSize)
    {
        int u_rep = find_repres(d_comp, i);
        for (int j = i + 1; j < V; ++j)
        {
            int v_rep = find_repres(d_comp, j);
            if (u_rep != v_rep)
            {
                weight_t w = sqrt(pow(d_xs[i] - d_xs[j], 2) + pow(d_ys[i] - d_ys[j], 2));
                ull weight_as_ull = static_cast<ull>(w * 1000.0);
                // Pack vertices u and v into the lower 32 bits. This assumes V < 65536.
                // A better approach for larger V would use two atomic operations or a 128-bit atomic.
                // For V=50000 this is fine. u is high 16 bits, v is low 16 bits.
                ull packed_indices = ((ull)i << 16) | j;
                ull packed_val = (weight_as_ull << 32) | packed_indices;
                atomicMin(&d_cheapest[u_rep], packed_val);
                atomicMin(&d_cheapest[v_rep], packed_val);
            }
        }
    }
}

__global__ void merge_kernel(int V, volatile int *d_comp, ull *d_cheapest, MSTEdge *d_mst_edges, int *d_mst_count, volatile bool *d_active)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int gridSize = gridDim.x * blockDim.x;

    for (int i = tid; i < V; i += gridSize)
    {
        if (d_comp[i] != i)
            continue; // Process only root nodes

        ull packed_val = d_cheapest[i];
        if (packed_val != ULL_INF)
        {
            ull packed_indices = packed_val & 0xFFFFFFFF;
            int u = packed_indices >> 16;
            int v = packed_indices & 0xFFFF;

            int u_rep = find_repres(d_comp, u);
            int v_rep = find_repres(d_comp, v);

            if (u_rep != v_rep)
            {
                int high_rep = max(u_rep, v_rep);
                int low_rep = min(u_rep, v_rep);
                int old_root = atomicCAS((int *)&d_comp[low_rep], low_rep, high_rep);

                if (old_root == low_rep)
                {
                    int mst_idx = atomicAdd(d_mst_count, 1);
                    d_mst_edges[mst_idx] = {u, v};
                    *d_active = true;
                }
            }
        }
    }
}

// Host function to orchestrate the CUDA Boruvka's MST
std::vector<std::vector<Edge>> BoruvkasAlgoCUDA(const VRP &vrp)
{
    int N = vrp.getSize();
    if (N <= 1)
        return std::vector<std::vector<Edge>>(N);

    // 1. Create coordinate lists on the host
    std::vector<point_t> h_xs(N), h_ys(N);
    for (int i = 0; i < N; ++i)
    {
        h_xs[i] = vrp.node[i].x;
        h_ys[i] = vrp.node[i].y;
    }

    // 2. Allocate GPU memory
    point_t *d_xs, *d_ys;
    MSTEdge *d_mst_edges;
    int *d_comp, *d_mst_count;
    ull *d_cheapest;
    bool *d_active;

    gpuErrchk(hipMalloc(&d_xs, N * sizeof(point_t)));
    gpuErrchk(hipMalloc(&d_ys, N * sizeof(point_t)));
    gpuErrchk(hipMalloc(&d_mst_edges, (N - 1) * sizeof(MSTEdge)));
    gpuErrchk(hipMalloc(&d_comp, N * sizeof(int)));
    gpuErrchk(hipMalloc(&d_cheapest, N * sizeof(ull)));
    gpuErrchk(hipMalloc(&d_mst_count, sizeof(int)));
    gpuErrchk(hipMalloc(&d_active, sizeof(bool)));

    // 3. Initialize and transfer data to GPU
    std::vector<int> h_comp(N);
    std::iota(h_comp.begin(), h_comp.end(), 0);

    gpuErrchk(hipMemcpy(d_xs, h_xs.data(), N * sizeof(point_t), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_ys, h_ys.data(), N * sizeof(point_t), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(d_comp, h_comp.data(), N * sizeof(int), hipMemcpyHostToDevice));
    gpuErrchk(hipMemset(d_mst_count, 0, sizeof(int)));

    // 4. Main loop on host, launching standard kernels
    bool h_active = true;
    while (h_active)
    {
        h_active = false;
        gpuErrchk(hipMemcpy(d_active, &h_active, sizeof(bool), hipMemcpyHostToDevice));

        int threadsPerBlock = 1024;
        int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;

        initialize_kernel<<<blocks, threadsPerBlock>>>(N, d_cheapest);
        gpuErrchk(hipGetLastError());

        find_cheapest_kernel<<<blocks, threadsPerBlock>>>(N, d_xs, d_ys, d_comp, d_cheapest);
        gpuErrchk(hipGetLastError());

        merge_kernel<<<blocks, threadsPerBlock>>>(N, d_comp, d_cheapest, d_mst_edges, d_mst_count, d_active);
        gpuErrchk(hipGetLastError());

        gpuErrchk(hipMemcpy(&h_active, d_active, sizeof(bool), hipMemcpyDeviceToHost));
        gpuErrchk(hipDeviceSynchronize());
    }

    // 5. Copy result back
    int h_mst_count;
    gpuErrchk(hipMemcpy(&h_mst_count, d_mst_count, sizeof(int), hipMemcpyDeviceToHost));
    std::vector<MSTEdge> h_mst_edges(h_mst_count);
    gpuErrchk(hipMemcpy(h_mst_edges.data(), d_mst_edges, h_mst_count * sizeof(MSTEdge), hipMemcpyDeviceToHost));

    // 6. Free memory
    gpuErrchk(hipFree(d_xs));
    gpuErrchk(hipFree(d_ys));
    gpuErrchk(hipFree(d_mst_edges));
    gpuErrchk(hipFree(d_comp));
    gpuErrchk(hipFree(d_cheapest));
    gpuErrchk(hipFree(d_mst_count));
    gpuErrchk(hipFree(d_active));

    // 7. Convert to adjacency list
    std::vector<std::vector<Edge>> nG(N);
    for (const auto &edge : h_mst_edges)
    {
        weight_t w = vrp.get_dist(edge.u, edge.v, vrp.params.toRound);
        nG[edge.u].push_back(Edge(edge.v, w));
        nG[edge.v].push_back(Edge(edge.u, w));
    }
    return nG;
}

// END: CUDA BORUVKA'S ALGORITHM IMPLEMENTATION

// THE REST OF THE VRP SOLVER CODE IS UNCHANGED
void ShortCircutTour(std::vector<std::vector<Edge>> &g, std::vector<bool> &visited, node_t u, std::vector<node_t> &out)
{
    visited[u] = true;
    out.push_back(u);
    for (auto e : g[u])
    {
        if (!visited[e.to])
        {
            ShortCircutTour(g, visited, e.to, out);
        }
    }
}

std::vector<std::vector<node_t>>
Split_convertToVrpRoutes(const VRP &vrp, const std::vector<node_t> &singleRoute)
{
    std::vector<node_t> customer_tour;
    customer_tour.reserve(vrp.size);
    for (node_t node : singleRoute)
    {
        if (node != DEPOT)
        {
            customer_tour.push_back(node);
        }
    }
    const int n = customer_tour.size();
    if (n == 0)
        return {};
    std::vector<double> sum_demands(n + 1, 0.0);
    std::vector<double> sum_dist(n + 1, 0.0);
    for (int i = 0; i < n; ++i)
    {
        sum_demands[i + 1] = sum_demands[i] + vrp.node[customer_tour[i]].demand;
        if (i > 0)
        {
            sum_dist[i + 1] = sum_dist[i] + vrp.get_dist(customer_tour[i - 1], customer_tour[i]);
        }
    }
    std::vector<weight_t> V(n + 1, std::numeric_limits<weight_t>::max());
    std::vector<int> P(n + 1, -1);
    V[0] = 0;
    std::deque<int> q;
    q.push_back(0);
    for (int j = 1; j <= n; ++j)
    {
        while (!q.empty() && sum_demands[j] - sum_demands[q.front()] > vrp.getCapacity())
        {
            q.pop_front();
        }
        auto calculate_total_cost = [&](int i)
        {
            double route_dist;
            if (i == j - 1)
            {
                route_dist = vrp.get_dist(DEPOT, customer_tour[i]) + vrp.get_dist(customer_tour[i], DEPOT);
            }
            else
            {
                route_dist = vrp.get_dist(DEPOT, customer_tour[i]) + (sum_dist[j] - sum_dist[i + 1]) + vrp.get_dist(customer_tour[j - 1], DEPOT);
            }
            return V[i] + route_dist;
        };
        while (q.size() >= 2)
        {
            if (calculate_total_cost(q[0]) >= calculate_total_cost(q[1]))
            {
                q.pop_front();
            }
            else
            {
                break;
            }
        }
        if (!q.empty())
        {
            P[j] = q.front();
            V[j] = calculate_total_cost(P[j]);
        }
        auto g = [&](int i)
        {
            if (i == 0)
                return 0.0;
            return V[i] - sum_dist[i] + vrp.get_dist(DEPOT, customer_tour[i - 1]);
        };
        while (!q.empty() && g(q.back()) >= g(j))
        {
            q.pop_back();
        }
        q.push_back(j);
    }
    std::vector<std::vector<node_t>> final_routes;
    int current_idx = n;
    while (current_idx > 0)
    {
        int pred_idx = P[current_idx];
        std::vector<node_t> new_route;
        for (int k = pred_idx; k < current_idx; ++k)
        {
            new_route.push_back(customer_tour[k]);
        }
        final_routes.push_back(new_route);
        current_idx = pred_idx;
    }
    std::reverse(final_routes.begin(), final_routes.end());
    return final_routes;
}

std::pair<weight_t, std::vector<std::vector<node_t>>>
calCost(const VRP &vrp, const std::vector<std::vector<node_t>> &final_routes)
{
    weight_t total_cost = 0.0;
    for (const auto &route : final_routes)
    {
        if (route.empty())
            continue;
        weight_t curr_route_cost = 0;
        curr_route_cost += vrp.get_dist(DEPOT, route[0]);
        for (unsigned jj = 1; jj < route.size(); ++jj)
        {
            curr_route_cost += vrp.get_dist(route[jj - 1], route[jj]);
        }
        curr_route_cost += vrp.get_dist(route.back(), DEPOT);
        total_cost += curr_route_cost;
    }
    return {total_cost, final_routes};
}

void tsp_2opt(const VRP &vrp, std::vector<node_t> &cities)
{
    if (cities.size() < 2)
        return;
    bool improved = true;
    while (improved)
    {
        improved = false;
        weight_t best_distance = calCost(vrp, {cities}).first;

        for (size_t i = 0; i < cities.size() - 1; i++)
        {
            for (size_t k = i + 1; k < cities.size(); k++)
            {
                std::vector<node_t> new_route = cities;
                std::reverse(new_route.begin() + i, new_route.begin() + k + 1);
                weight_t new_distance = calCost(vrp, {new_route}).first;
                if (new_distance < best_distance)
                {
                    cities = new_route;
                    best_distance = new_distance;
                    improved = true;
                }
            }
        }
    }
}

std::vector<std::vector<node_t>>
postProcessIt(const VRP &vrp, std::vector<std::vector<node_t>> &final_routes, weight_t &minCost)
{
    std::vector<std::vector<node_t>> postprocessed_routes = final_routes;
#pragma omp parallel for
    for (unsigned i = 0; i < postprocessed_routes.size(); ++i)
    {
        tsp_2opt(vrp, postprocessed_routes[i]);
    }
    minCost = calCost(vrp, postprocessed_routes).first;
    return postprocessed_routes;
}

bool verify_sol(const VRP &vrp, const vector<vector<node_t>> &final_routes, unsigned capacity)
{
    std::vector<bool> visited(vrp.getSize(), false);
    visited[DEPOT] = true;
    for (const auto &route : final_routes)
    {
        demand_t route_demand = 0;
        for (node_t node : route)
        {
            if (visited[node])
                return false; // Visited twice
            visited[node] = true;
            route_demand += vrp.node[node].demand;
        }
        if (route_demand > capacity)
            return false; // Exceeds capacity
    }
    for (size_t i = 1; i < vrp.getSize(); ++i)
    {
        if (!visited[i])
            return false; // Not all customers visited
    }
    return true;
}

// MAIN
int main(int argc, char *argv[])
{
    VRP vrp;
    if (argc < 2)
    {
        std::cout << "parMDS version 1.1 (Boruvka's MST)" << '\n';
        std::cout << "Usage: " << argv[0] << " toy.vrp [-nthreads <n> DEFAULT is 20] [-round 0 or 1 DEFAULT:1]" << '\n';
        exit(1);
    }

    for (int ii = 2; ii < argc; ii += 2)
    {
        if (std::string(argv[ii]) == "-round")
            vrp.params.toRound = atoi(argv[ii + 1]);
        else if (std::string(argv[ii]) == "-nthreads")
            vrp.params.nThreads = atoi(argv[ii + 1]);
        else
        {
            std::cerr << "INVALID Arguments!" << '\n';
            std::cerr << "Usage:" << argv[0] << " toy.vrp -nthreads 20 -round 1" << '\n';
            exit(1);
        }
    }

    vrp.read(argv[1]);

    std::chrono::high_resolution_clock::time_point start = std::chrono::high_resolution_clock::now();

    // =========================================================================
    // MODIFICATION: Replace Prim's with Boruvka's Algorithm
    // =========================================================================
    auto mstG = BoruvkasAlgoCUDA(vrp);
    // =========================================================================

    std::vector<bool> visited(mstG.size(), false);
    visited[0] = true;
    std::vector<int> singleRoute;

    weight_t minCost = DBL_MAX;
    std::vector<std::vector<node_t>> minRoute;

    auto mstCopy = mstG;

    for (int i = 0; i < 1; i++)
    {
        for (auto &list : mstCopy)
        {
            std::shuffle(list.begin(), list.end(), std::default_random_engine(0));
        }

        std::vector<int> singleRoute;

        std::vector<bool> visited(mstCopy.size(), false);
        visited[0] = true;

        ShortCircutTour(mstCopy, visited, 0, singleRoute);
        DEBUG std::cout << '\n';

        auto aRoutes = Split_convertToVrpRoutes(vrp, singleRoute);

        auto aCostRoute = calCost(vrp, aRoutes);
        if (aCostRoute.first < minCost)
        {
            minCost = aCostRoute.first;
            minRoute = aCostRoute.second;
        }
    }

    auto minCost1 = minCost;

    std::chrono::high_resolution_clock::time_point end = std::chrono::high_resolution_clock::now();
    uint64_t elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

    auto timeUpto1 = (double)(elapsed * 1.E-9);
    short PARLIMIT = vrp.params.nThreads;
    std::chrono::high_resolution_clock::time_point start2 = std::chrono::high_resolution_clock::now();

#pragma omp parallel for shared(minCost, minRoute) num_threads(PARLIMIT)
    for (int i = 0; i < 100000; i += PARLIMIT)
    {
        for (auto &list : mstCopy)
        {
            std::shuffle(list.begin(), list.end(), std::default_random_engine(rand()));
        }

        std::vector<int> singleRoute;
        std::vector<bool> visited(mstCopy.size(), false);
        visited[0] = true;

        ShortCircutTour(mstCopy, visited, 0, singleRoute);
        DEBUG std::cout << '\n';

        auto aRoutes = Split_convertToVrpRoutes(vrp, singleRoute);

        auto aCostRoute = calCost(vrp, aRoutes);
        if (aCostRoute.first < minCost)
        {
            minCost = aCostRoute.first;
            minRoute = aCostRoute.second;
        }
    }

    auto minCost2 = minCost;
    end = std::chrono::high_resolution_clock::now();
    elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start2).count();

    auto timeUpto2 = (double)(elapsed * 1.E-9);
    std::chrono::high_resolution_clock::time_point start3 = std::chrono::high_resolution_clock::now();

    auto postRoutes = postProcessIt(vrp, minRoute, minCost);

    end = std::chrono::high_resolution_clock::now();
    elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start3).count();
    double timeUpto3 = (double)(elapsed * 1.E-9);
    elapsed = std::chrono::duration_cast<std::chrono::nanoseconds>(end - start).count();

    double total_time = (double)(elapsed * 1.E-9);

    bool verified = false;
    verified = verify_sol(vrp, postRoutes, vrp.getCapacity());

    std::cout << argv[1] << " Cost ";
    std::cout << "Pre-Refine COST = ";
    std::cout << minCost2 << ',';
    std::cout << "Final Cost = ";
    std::cout << minCost;

    std::cout << " | Time(s): ";
    std::cout << "MST = ";
    std::cout << timeUpto1 << ',';
    std::cout << "Refinement = ";
    std::cout << timeUpto2 << ',';
    std::cout << "Post-Processing = ";
    std::cout << timeUpto3 << ",";
    std::cout << "Total = ";
    std::cout << total_time;

    if (verified)
        std::cout << " VALID" << std::endl;
    else
        std::cout << " INVALID" << std::endl;

    return 0;
}