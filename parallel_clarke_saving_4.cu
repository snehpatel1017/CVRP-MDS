#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cmath>
#include <cfloat>
#include <iostream>
#include <vector>
#include <string>
#include <fstream>
#include <sstream>
#include <cmath>
#include <algorithm>
#include <cfloat>  // For DBL_MAX
#include <iomanip> // For std::setprecision
#include <chrono>  // For timing
#include <omp.h>
// For DBL_MAX

// CUDA specific headers
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_cooperative_groups.h> // Required for grid.sync()

namespace cg = cooperative_groups;

using namespace std;

using point_t = double;
using weight_t = double;
using demand_t = double;
using node_t = int;

const node_t DEPOT = 0;

struct Point
{
    double x, y, demand;
};

struct Saving
{
    volatile int i, j;
    volatile double value;
};

class VRP
{
public:
    size_t size;
    demand_t capacity;
    std::vector<Point> node;
    std::vector<weight_t> dist_to_depot;

    VRP() : size(0), capacity(0) {}

    void read(const std::string &filename);
    weight_t get_dist(node_t i, node_t j) const;

    size_t getSize() const
    {
        return size;
    }
    demand_t getCapacity() const
    {
        return capacity;
    }
};

void VRP::read(const std::string &filename)
{
    std::ifstream in(filename);
    if (!in.is_open())
    {
        std::cerr << "Error: Could not open file " << filename << std::endl;
        exit(1);
    }
    std::string line;
    while (getline(in, line) && line.find("DIMENSION") == std::string::npos)
        ;
    if (line.find(":") != std::string::npos)
        size = stoul(line.substr(line.find(":") + 1));
    while (getline(in, line) && line.find("CAPACITY") == std::string::npos)
        ;
    if (line.find(":") != std::string::npos)
        capacity = stoul(line.substr(line.find(":") + 1));
    while (getline(in, line) && line.find("NODE_COORD_SECTION") == std::string::npos)
        ;
    node.resize(size);
    for (size_t i = 0; i < size; ++i)
    {
        int id;
        in >> id >> node[i].x >> node[i].y;
    }
    while (getline(in, line) && line.find("DEMAND_SECTION") == std::string::npos)
        ;
    for (size_t i = 0; i < size; ++i)
    {
        int id;
        in >> id >> node[i].demand;
    }
    in.close();
    dist_to_depot.resize(size);
    for (size_t i = 0; i < size; ++i)
    {
        dist_to_depot[i] = get_dist(DEPOT, i);
    }
}

weight_t VRP::get_dist(node_t i, node_t j) const
{
    double dx = node[i].x - node[j].x;
    double dy = node[i].y - node[j].y;
    return sqrt(dx * dx + dy * dy);
}

weight_t calCost(const VRP &vrp, const std::vector<std::vector<node_t>> &routes)
{
    weight_t total_cost = 0.0;
    for (const auto &route : routes)
    {
        if (route.empty())
            continue;

        node_t last_node = DEPOT;
        for (node_t current_node : route)
        {
            total_cost += vrp.get_dist(last_node, current_node);
            last_node = current_node;
        }
        // Add cost to return to the depot
        total_cost += vrp.get_dist(last_node, DEPOT);
    }
    return total_cost;
}

bool verify_sol(const VRP &vrp, const std::vector<std::vector<node_t>> &routes)
{
    for (const auto &route : routes)
    {
        demand_t route_demand = 0;
        for (node_t customer : route)
        {
            if (customer < 0 || customer >= vrp.size)
                return false; // Invalid node
            route_demand += vrp.node[customer].demand;
        }
        if (route_demand > vrp.getCapacity())
        {
            return false; // Capacity violated
        }
    }
    return true;
}

#define checkCudaErrors(val) check_cuda((val), #val, __FILE__, __LINE__)
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line)
{
    if (result)
    {
        std::cerr << "CUDA Error at " << file << ":" << line << " code=" << static_cast<unsigned int>(result)
                  << " \"" << hipGetErrorString(result) << "\" for " << func << std::endl;
        hipDeviceReset();
        exit(99);
    }
}

__device__ double device_euclidean_dist(const Point &a, const Point &b)
{
    return sqrt((a.x - b.x) * (a.x - b.x) + (a.y - b.y) * (a.y - b.y));
}

__global__ void find_best_saving_kernel(
    const Point *nodes,
    const node_t *customer_route_map,
    const demand_t *route_demands,
    const node_t *route_head,
    const node_t *route_tail,
    const weight_t *dist_to_depot,
    Saving *best_saving_out,
    int num_customers,
    demand_t capacity)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;

    if (i >= (num_customers + 1) || j >= (num_customers + 1) || i >= j)
    {
        return;
    }

    // --- 2. Check validity of merging pair (i, j) ---
    node_t route_id_i = customer_route_map[i];
    node_t route_id_j = customer_route_map[j];

    if (route_id_i == route_id_j || route_id_i < 0 || route_id_j < 0)
        return;
    if (route_demands[route_id_i] + route_demands[route_id_j] > capacity)
        return;

    node_t front_i = route_head[route_id_i];
    node_t back_i = route_tail[route_id_i];
    node_t front_j = route_head[route_id_j];
    node_t back_j = route_tail[route_id_j];

    if (!((i == front_i || i == back_i) && (j == front_j || j == back_j)))
    {
        return;
    }

    // --- 3. Calculate saving if the merge is valid ---
    weight_t saving_value = dist_to_depot[i]                             // dist(i, depot)
                            + dist_to_depot[j]                           // dist(j, depot)
                            - device_euclidean_dist(nodes[i], nodes[j]); // dist(i, j)

    unsigned long long int *address_as_ull = (unsigned long long int *)&(best_saving_out->value);

    // Read the current maximum value from global memory.
    double current_max_val = __longlong_as_double(*address_as_ull);

    // This loop continues as long as this thread's saving is better than the global max.
    while (saving_value > current_max_val)
    {
        // Convert our local values to their bit representations for the atomic operation.
        unsigned long long int assumed_ull = __double_as_longlong(current_max_val);
        unsigned long long int new_val_ull = __double_as_longlong(saving_value);

        unsigned long long int prev_val_ull = atomicCAS(address_as_ull, assumed_ull, new_val_ull);

        if (prev_val_ull == assumed_ull)
        {
            // Now this thread has the exclusive right to update the i and j indices.
            best_saving_out->i = i;
            best_saving_out->j = j;
            break; // Success, exit the loop.
        }

        current_max_val = __longlong_as_double(prev_val_ull);
    }
}

__global__ void update_gpu_mempory(
    int type,
    node_t *customer_route_map,
    demand_t *route_demands,
    node_t *route_head,
    node_t *route_tail,
    node_t *next_customer,
    node_t *prev_customer,
    node_t *temporary,
    int num_customers,
    node_t i,
    node_t j,
    node_t route_id_i,
    node_t route_id_j,
    node_t head_i,
    node_t tail_i,
    node_t head_j,
    node_t tail_j)
{
    cg::grid_group grid = cg::this_grid();
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int total_threads = blockDim.x * gridDim.x;
    if (tid == 0)
    {
        if (type == 1)
        {
            next_customer[i] = j;
            prev_customer[j] = i;
            route_tail[route_id_i] = tail_j;
        }
        else if (type == 2)
        {
            next_customer[j] = i;
            prev_customer[i] = j;
            route_head[route_id_i] = head_j;
        }
        else if (type == 3)
        {
            next_customer[j] = prev_customer[j];
            next_customer[i] = j;
            prev_customer[j] = i;
            route_tail[route_id_i] = head_j;
            customer_route_map[j] = route_id_i;
        }
        else
        {
            prev_customer[i] = next_customer[i];
            next_customer[j] = i;
            prev_customer[i] = j;
            route_head[route_id_i] = tail_j;
        }

        route_demands[route_id_i] += route_demands[route_id_j];
    }
    grid.sync();

    if (type == 3)
    {
        for (node_t curr = tid; curr < num_customers; curr += total_threads)
        {
            if (customer_route_map[curr] == route_id_j)
            {
                temporary[curr] = prev_customer[curr];
            }
        }
        grid.sync();

        for (node_t curr = tid; curr < num_customers; curr += total_threads)
        {
            if (customer_route_map[curr] == route_id_j)
            {
                prev_customer[curr] = next_customer[curr];
                next_customer[curr] = temporary[curr];
            }
        }
    }
    else if (type == 4)
    {
        for (node_t curr = tid; curr < num_customers; curr += total_threads)
        {
            if (curr == i)
                continue;
            if (customer_route_map[curr] == route_id_i)
            {
                temporary[curr] = prev_customer[curr];
            }
        }
        grid.sync();
        for (node_t curr = tid; curr < num_customers; curr += total_threads)
        {
            if (curr == i)
                continue;
            if (customer_route_map[curr] == route_id_i)
            {
                prev_customer[curr] = next_customer[curr];
                next_customer[curr] = temporary[curr];
                customer_route_map[curr] = route_id_i;
            }
        }
    }
    grid.sync();
    for (node_t curr = tid; curr < num_customers; curr += total_threads)
    {
        if (customer_route_map[curr] == route_id_j)
        {
            customer_route_map[curr] = route_id_i;
        }
    }
    if (tid == 0)
    {
        route_head[route_id_j] = DEPOT;
    }
}

std::vector<std::vector<node_t>> parallel_savings_algorithm(const VRP &vrp)
{
    const int NUM_CUSTOMERS = vrp.getSize() - 1; // Exclude depot
    const demand_t CAPACITY = vrp.getCapacity();

    std::vector<node_t> h_customer_route_map(NUM_CUSTOMERS + 1);
    std::vector<demand_t> h_route_demands(NUM_CUSTOMERS + 1);
    std::vector<node_t> h_route_head(NUM_CUSTOMERS + 1);
    std::vector<node_t> h_route_tail(NUM_CUSTOMERS + 1);
    std::vector<node_t> h_next_customer(vrp.size, DEPOT);
    std::vector<node_t> h_prev_customer(vrp.size, DEPOT);

    for (int i = 1; i <= NUM_CUSTOMERS; ++i)
    {
        // Initially, each customer is in their own route
        h_customer_route_map[i] = i;
        h_route_demands[i] = vrp.node[i].demand;
        // The start and end of the route is just the customer itself
        h_route_head[i] = i;
        h_route_tail[i] = i;
    }

    // --- 2. DEVICE: Allocate GPU memory ---
    Point *d_nodes;
    node_t *d_customer_route_map;
    demand_t *d_route_demands;
    node_t *d_route_head;
    node_t *d_route_tail;
    node_t *d_next_customer;
    node_t *d_prev_customer;
    node_t *d_temporary;
    Saving *d_best_saving_out;
    weight_t *d_dist_to_depot;

    checkCudaErrors(hipMalloc(&d_nodes, (NUM_CUSTOMERS + 1) * sizeof(Point)));
    checkCudaErrors(hipMalloc(&d_customer_route_map, (NUM_CUSTOMERS + 1) * sizeof(node_t)));
    checkCudaErrors(hipMalloc(&d_route_demands, (NUM_CUSTOMERS + 1) * sizeof(demand_t)));
    checkCudaErrors(hipMalloc(&d_route_head, (NUM_CUSTOMERS + 1) * sizeof(node_t)));
    checkCudaErrors(hipMalloc(&d_route_tail, (NUM_CUSTOMERS + 1) * sizeof(node_t)));
    checkCudaErrors(hipMalloc(&d_next_customer, vrp.size * sizeof(node_t)));
    checkCudaErrors(hipMalloc(&d_prev_customer, vrp.size * sizeof(node_t)));
    checkCudaErrors(hipMalloc(&d_temporary, vrp.size * sizeof(node_t)));
    checkCudaErrors(hipMalloc(&d_best_saving_out, sizeof(Saving)));
    checkCudaErrors(hipMalloc(&d_dist_to_depot, (NUM_CUSTOMERS + 1) * sizeof(weight_t)));

    // --- 3. HOST -> DEVICE: Copy data to GPU ---
    checkCudaErrors(hipMemcpy(d_nodes, vrp.node.data(), (NUM_CUSTOMERS + 1) * sizeof(Point), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_customer_route_map, h_customer_route_map.data(), (NUM_CUSTOMERS + 1) * sizeof(node_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_route_demands, h_route_demands.data(), (NUM_CUSTOMERS + 1) * sizeof(demand_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_route_head, h_route_head.data(), (NUM_CUSTOMERS + 1) * sizeof(node_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_route_tail, h_route_tail.data(), (NUM_CUSTOMERS + 1) * sizeof(node_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_next_customer, h_next_customer.data(), vrp.size * sizeof(node_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_prev_customer, h_prev_customer.data(), vrp.size * sizeof(node_t), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_dist_to_depot, vrp.dist_to_depot.data(), (NUM_CUSTOMERS + 1) * sizeof(weight_t), hipMemcpyHostToDevice));
    // Initialize the output struct on the GPU to a known "worst" state
    Saving h_best_saving_init = {-1, -1, -DBL_MAX};
    checkCudaErrors(hipMemcpy(d_best_saving_out, &h_best_saving_init, sizeof(Saving), hipMemcpyHostToDevice));

    // --- 4. KERNEL LAUNCH ---
    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks(
        (NUM_CUSTOMERS + 1 + threadsPerBlock.x - 1) / threadsPerBlock.x,
        (NUM_CUSTOMERS + 1 + threadsPerBlock.y - 1) / threadsPerBlock.y);

    int threads_per_block = std::min(1024, NUM_CUSTOMERS);
    int numBlocks1D = std::min(30, (NUM_CUSTOMERS + threads_per_block - 1) / threads_per_block);

    Saving h_result;

    // --- 4. Merge Routes Greedily (Sequential) ---
    int id = 0;
    while (true)
    {
        // std::cout << id++ << "\n";

        id++;
        checkCudaErrors(hipMemcpy(d_best_saving_out, &h_best_saving_init, sizeof(Saving), hipMemcpyHostToDevice));
        find_best_saving_kernel<<<numBlocks, threadsPerBlock>>>(
            d_nodes, d_customer_route_map, d_route_demands, d_route_head, d_route_tail, d_dist_to_depot,
            d_best_saving_out, NUM_CUSTOMERS, CAPACITY);
        checkCudaErrors(hipDeviceSynchronize());

        checkCudaErrors(hipMemcpy(&h_result, d_best_saving_out, sizeof(Saving), hipMemcpyDeviceToHost));
        if (h_result.value <= 1e-6)
        {
            std::cout << "No more positive savings found. Halting." << std::endl;
            std::cout << id << "\n";
            break; // Exit the while loop
        }

        node_t i = h_result.i;
        node_t j = h_result.j;
        std::cout << i << " " << j << "\n";

        node_t route_id_i = h_customer_route_map[i];
        node_t route_id_j = h_customer_route_map[j];

        // Check if the merge is valid (different routes and combined demand is within capacity)
        if (route_id_i != route_id_j && h_route_demands[route_id_i] + h_route_demands[route_id_j] <= vrp.capacity)
        {
            node_t head_i = h_route_head[route_id_i];
            node_t tail_i = h_route_tail[route_id_i];
            node_t head_j = h_route_head[route_id_j];
            node_t tail_j = h_route_tail[route_id_j];

            bool merged = false;
            int type = -1;

            if (tail_i == i && head_j == j)
            {

                h_route_tail[route_id_i] = tail_j; // New tail is old tail of j
                merged = true;
                type = 1;
            }
            // Case 2: Tail of route j connects to Head of route i [...j] -> [i...]
            else if (tail_j == j && head_i == i)
            {

                h_route_head[route_id_i] = head_j; // New head is old head of j
                merged = true;
                type = 2;
            }
            // Case 3: Tail of i connects to Tail of j [...i] -> [...j](reversed)
            else if (tail_i == i && tail_j == j)
            {

                h_route_tail[route_id_i] = head_j; // New tail is old head of j
                merged = true;
                type = 3;
            }
            // Case 4: Head of i connects to Head of j [i...](reversed) <- [j...]
            else if (head_i == i && head_j == j)
            {

                h_route_head[route_id_i] = tail_j; // New head is old tail of j
                merged = true;
                type = 4;
            }

            if (merged)
            {

                h_route_demands[route_id_i] += h_route_demands[route_id_j];
                h_route_demands[route_id_j] = 0;
                h_customer_route_map[j] = route_id_i;
                h_route_head[route_id_j] = DEPOT;
                void *args[] = {
                    (void *)&type,
                    &d_customer_route_map,
                    &d_route_demands,
                    &d_route_head,
                    &d_route_tail,
                    &d_next_customer,
                    &d_prev_customer,
                    &d_temporary,
                    (void *)&NUM_CUSTOMERS,
                    (void *)&i,
                    (void *)&j,
                    (void *)&route_id_i,
                    (void *)&route_id_j,
                    (void *)&head_i,
                    (void *)&tail_i,
                    (void *)&head_j,
                    (void *)&tail_j};

                checkCudaErrors(hipLaunchCooperativeKernel(
                    (void *)update_gpu_mempory,
                    numBlocks1D,
                    threads_per_block,
                    args));
                checkCudaErrors(hipDeviceSynchronize());
            }
        }
    }
    std::cout << "loop ended\n";

    checkCudaErrors(hipMemcpy(h_customer_route_map.data(), d_customer_route_map, (NUM_CUSTOMERS + 1) * sizeof(node_t), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_route_demands.data(), d_route_demands, (NUM_CUSTOMERS + 1) * sizeof(demand_t), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_route_head.data(), d_route_head, (NUM_CUSTOMERS + 1) * sizeof(node_t), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_route_tail.data(), d_route_tail, (NUM_CUSTOMERS + 1) * sizeof(node_t), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_next_customer.data(), d_next_customer, vrp.size * sizeof(node_t), hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(h_prev_customer.data(), d_prev_customer, vrp.size * sizeof(node_t), hipMemcpyDeviceToHost));

    // --- 5. Finalize Routes ---
    std::vector<std::vector<node_t>> final_routes;
    std::vector<bool> visited_routes(vrp.size, false);

    for (node_t i = 1; i < vrp.size; ++i)
    {
        node_t route_id = h_customer_route_map[i];
        if (route_id != DEPOT && !visited_routes[route_id])
        {
            visited_routes[route_id] = true;
            std::vector<node_t> current_route;
            node_t current_node = h_route_head[route_id];
            while (current_node != DEPOT)
            {
                current_route.push_back(current_node);
                current_node = h_next_customer[current_node];
            }
            if (!current_route.empty())
            {
                final_routes.push_back(current_route);
            }
        }
    }

    checkCudaErrors(hipFree(d_nodes));
    checkCudaErrors(hipFree(d_customer_route_map));
    checkCudaErrors(hipFree(d_route_demands));
    checkCudaErrors(hipFree(d_best_saving_out));
    checkCudaErrors(hipFree(d_dist_to_depot));
    checkCudaErrors(hipFree(d_route_head));
    checkCudaErrors(hipFree(d_route_tail));
    checkCudaErrors(hipFree(d_next_customer));
    checkCudaErrors(hipFree(d_prev_customer));
    checkCudaErrors(hipFree(d_temporary));
    checkCudaErrors(hipDeviceReset());

    return final_routes;
}

void tsp_approx(const VRP &vrp, std::vector<node_t> &cities, std::vector<node_t> &tour, node_t ncities)
{
    node_t i, j;
    node_t ClosePt = 0;
    weight_t CloseDist;

    for (i = 1; i < ncities; i++)
        tour[i] = cities[i - 1];

    tour[0] = cities[ncities - 1];

    for (i = 1; i < ncities; i++)
    {
        weight_t ThisX = vrp.node[tour[i - 1]].x;
        weight_t ThisY = vrp.node[tour[i - 1]].y;
        CloseDist = DBL_MAX;
        for (j = ncities - 1;; j--)
        {
            weight_t ThisDist = (vrp.node[tour[j]].x - ThisX) * (vrp.node[tour[j]].x - ThisX);
            if (ThisDist <= CloseDist)
            {
                ThisDist += (vrp.node[tour[j]].y - ThisY) * (vrp.node[tour[j]].y - ThisY);
                if (ThisDist <= CloseDist)
                {
                    if (j < i)
                        break;
                    CloseDist = ThisDist;
                    ClosePt = j;
                }
            }
        }
        unsigned temp = tour[i];
        tour[i] = tour[ClosePt];
        tour[ClosePt] = temp;
    }
}

std::vector<std::vector<node_t>>
postprocess_tsp_approx(const VRP &vrp, std::vector<std::vector<node_t>> &solRoutes)
{
    std::vector<std::vector<node_t>> modifiedRoutes;

    unsigned nroutes = solRoutes.size();
    for (unsigned i = 0; i < nroutes; ++i)
    {
        unsigned sz = solRoutes[i].size();
        std::vector<node_t> cities(sz + 1);
        std::vector<node_t> tour(sz + 1);

        for (unsigned j = 0; j < sz; ++j)
            cities[j] = solRoutes[i][j];

        cities[sz] = 0;

        tsp_approx(vrp, cities, tour, sz + 1);

        vector<node_t> curr_route;
        for (unsigned kk = 1; kk < sz + 1; ++kk)
        {
            curr_route.push_back(tour[kk]);
        }

        modifiedRoutes.push_back(curr_route);
    }
    return modifiedRoutes;
}

void tsp_2opt(const VRP &vrp, std::vector<node_t> &cities, std::vector<node_t> &tour, unsigned ncities)
{
    unsigned improve = 0;

    while (improve < 2)
    {
        double best_distance = 0.0;

        best_distance += vrp.get_dist(DEPOT, cities[0]);

        for (unsigned jj = 1; jj < ncities; ++jj)
        {
            best_distance += vrp.get_dist(cities[jj - 1], cities[jj]);
        }

        best_distance += vrp.get_dist(DEPOT, cities[ncities - 1]);

        for (unsigned i = 0; i < ncities - 1; i++)
        {
            for (unsigned k = i + 1; k < ncities; k++)
            {
                double new_distance = best_distance;
                if (i == 0)
                    new_distance -= vrp.get_dist(DEPOT, cities[i]);
                else
                    new_distance -= vrp.get_dist(cities[i - 1], cities[i]);

                if (k == ncities - 1)
                    new_distance -= vrp.get_dist(cities[k], DEPOT);
                else
                    new_distance -= vrp.get_dist(cities[k], cities[k + 1]);

                if (i == 0)
                    new_distance += vrp.get_dist(DEPOT, cities[k]);
                else
                    new_distance += vrp.get_dist(cities[i - 1], cities[k]);

                if (k == ncities - 1)
                    new_distance += vrp.get_dist(cities[i], DEPOT);
                else
                    new_distance += vrp.get_dist(cities[i], cities[k + 1]);

                if (new_distance < best_distance)
                {
                    improve = 0;
                    int left_ptr = i, right_ptr = k;
                    while (left_ptr <= right_ptr)
                    {
                        swap(cities[left_ptr++], cities[right_ptr--]);
                    }
                    best_distance = new_distance;
                }
            }
        }
        improve++;
    }
}
std::vector<std::vector<node_t>>
postprocess_2OPT(const VRP &vrp, std::vector<std::vector<node_t>> &final_routes)
{
    std::vector<std::vector<node_t>> postprocessed_final_routes;

    unsigned nroutes = final_routes.size();
    for (unsigned i = 0; i < nroutes; ++i)
    {
        unsigned sz = final_routes[i].size();
        std::vector<node_t> cities(sz);
        std::vector<node_t> tour(sz);

        for (unsigned j = 0; j < sz; ++j)
            cities[j] = final_routes[i][j];

        vector<node_t> curr_route;

        if (sz > 2)
            tsp_2opt(vrp, cities, tour, sz);

        for (unsigned kk = 0; kk < sz; ++kk)
        {
            curr_route.push_back(cities[kk]);
        }

        postprocessed_final_routes.push_back(curr_route);
    }
    return postprocessed_final_routes;
}

weight_t get_total_cost_of_routes(const VRP &vrp, vector<vector<node_t>> &final_routes)
{
    weight_t total_cost = 0.0;
    for (unsigned ii = 0; ii < final_routes.size(); ++ii)
    {
        weight_t curr_route_cost = 0;
        curr_route_cost += vrp.get_dist(DEPOT, final_routes[ii][0]);
        for (unsigned jj = 1; jj < final_routes[ii].size(); ++jj)
        {
            curr_route_cost += vrp.get_dist(final_routes[ii][jj - 1], final_routes[ii][jj]);
        }
        curr_route_cost += vrp.get_dist(DEPOT, final_routes[ii][final_routes[ii].size() - 1]);

        total_cost += curr_route_cost;
    }

    return total_cost;
}

std::vector<std::vector<node_t>>
postProcessIt(const VRP &vrp, std::vector<std::vector<node_t>> &final_routes, weight_t &minCost)
{
    std::vector<std::vector<node_t>> postprocessed_final_routes;

    auto postprocessed_final_routes1 = postprocess_tsp_approx(vrp, final_routes);
    auto postprocessed_final_routes2 = postprocess_2OPT(vrp, postprocessed_final_routes1);
    auto postprocessed_final_routes3 = postprocess_2OPT(vrp, final_routes);

#pragma omp parallel for
    for (unsigned zzz = 0; zzz < final_routes.size(); ++zzz)
    {
        vector<node_t> postprocessed_route2 = postprocessed_final_routes2[zzz];
        vector<node_t> postprocessed_route3 = postprocessed_final_routes3[zzz];

        unsigned sz2 = postprocessed_route2.size();
        unsigned sz3 = postprocessed_route3.size();

        weight_t postprocessed_route2_cost = 0.0;
        postprocessed_route2_cost += vrp.get_dist(DEPOT, postprocessed_route2[0]);
        for (unsigned jj = 1; jj < sz2; ++jj)
        {
            postprocessed_route2_cost += vrp.get_dist(postprocessed_route2[jj - 1], postprocessed_route2[jj]);
        }
        postprocessed_route2_cost += vrp.get_dist(DEPOT, postprocessed_route2[sz2 - 1]);

        weight_t postprocessed_route3_cost = 0.0;
        postprocessed_route3_cost += vrp.get_dist(DEPOT, postprocessed_route3[0]);
        for (unsigned jj = 1; jj < sz3; ++jj)
        {
            postprocessed_route3_cost += vrp.get_dist(postprocessed_route3[jj - 1], postprocessed_route3[jj]);
        }
        postprocessed_route3_cost += vrp.get_dist(DEPOT, postprocessed_route3[sz3 - 1]);

        if (postprocessed_route3_cost > postprocessed_route2_cost)
        {
            postprocessed_final_routes.push_back(postprocessed_route2);
        }
        else
        {
            postprocessed_final_routes.push_back(postprocessed_route3);
        }
    }

    auto postprocessed_final_routes_cost = get_total_cost_of_routes(vrp, postprocessed_final_routes);

    minCost = postprocessed_final_routes_cost;

    return postprocessed_final_routes;
}

int main(int argc, char *argv[])
{
    if (argc < 2)
    {
        std::cerr << "Usage: " << argv[0] << " <filename.vrp> [num_threads]" << std::endl;
        return 1;
    }

    VRP vrp;
    vrp.read(argv[1]);

    auto start_time = std::chrono::high_resolution_clock::now();
    std::vector<std::vector<node_t>> routes = parallel_savings_algorithm(vrp);
    auto end_time = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = end_time - start_time;
    weight_t total_cost = calCost(vrp, routes);
    auto postRoutes = postProcessIt(vrp, routes, total_cost);
    total_cost = calCost(vrp, postRoutes);
    bool is_valid = verify_sol(vrp, postRoutes);

    std::cout << "--- Parallel Clarke & Wright Savings Algorithm ---" << std::endl;
    std::cout << "Problem File: " << argv[1] << std::endl;
    // std::cout << "Threads Used: " << omp_get_max_threads() << std::endl;
    std::cout << "--------------------------------------------------" << std::endl;
    std::cout << std::fixed << std::setprecision(2);
    std::cout << "Total Solution Cost: " << total_cost << std::endl;
    std::cout << "Number of Routes:   " << postRoutes.size() << std::endl;
    std::cout << "Total Time Taken:    " << elapsed.count() << " seconds" << std::endl;
    std::cout << "Solution Validity:   " << (is_valid ? "VALID" : "INVALID") << std::endl;
    std::cout << "--------------------------------------------------" << std::endl;

    return 0;
}
